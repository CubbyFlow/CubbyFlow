#include "hip/hip_runtime.h"

//--------------------------------------------------------------------------------
// NVIDIA(R) GVDB VOXELS
// Copyright 2017, NVIDIA Corporation
//
// Redistribution and use in source and binary forms, with or without modification, 
// are permitted provided that the following conditions are met:
// 1. Redistributions of source code must retain the above copyright notice, this 
//    list of conditions and the following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice, this 
//    list of conditions and the following disclaimer in the documentation and/or 
//    other materials provided with the distribution.
// 3. Neither the name of the copyright holder nor the names of its contributors may 
//    be used to endorse or promote products derived from this software without specific 
//   prior written permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY 
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES 
// OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT 
// SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, 
// SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT 
// OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) 
// HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR 
// TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, 
// EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
// 
// Version 1.0: Rama Hoetzlein, 5/1/2017
//----------------------------------------------------------------------------------


#include "optix_extra_math.cuh"
#include ""			// from OptiX SDK

struct PerRayData_radiance
{
	float3	result;
	float	length; 
	float	alpha;
	int		depth;
	int		rtype;
};

rtDeclareVariable(float3,        cam_pos, , );
rtDeclareVariable(float3,		 cam_U, , );
rtDeclareVariable(float3,		 cam_V, , );
rtDeclareVariable(float3,		 cam_S, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(unsigned int,  sample, , );
rtBuffer<float3, 2>              output_buffer;
rtDeclareVariable(rtObject,      top_object, , );
rtBuffer<unsigned int, 2>        rnd_seeds;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

float3 __device__ __inline__ jitter_sample ()
{	 
	uint2 index = make_uint2(launch_index.x % 128, launch_index.y % 128);
    unsigned int seed  = rnd_seeds[ index ]; 
    float uu = rnd( seed )-0.5f;
    float vv = rnd( seed )-0.5f;
	float ww = rnd( seed )-0.5f;    
    return make_float3(uu,vv,ww);
}


__device__ float3 getViewRay(float x, float y, float pixsize)
{
	float3 v = x*cam_U + y*cam_V + cam_S + jitter_sample()*make_float3(pixsize, pixsize, pixsize);
	return normalize(v);
}

RT_PROGRAM void trace_primary ()
{
  float2 d = make_float2(launch_index) / make_float2(launch_dim);
  float pixsize = length ( cam_U ) / launch_dim.x;	
  float3 ray_direction;
  float3 result;

  PerRayData_radiance prd;
  prd.length = 0.f;
  prd.alpha = 1.f;
  prd.depth = 0;
  prd.rtype = 0;	// ANY_RAY

  int initial_samples = 1;

  if ( sample <= initial_samples ) {
	  result = make_float3(0,0,0);	  
	  for (int n=0; n < initial_samples; n++ ) {
		  ray_direction = getViewRay(d.x, d.y, pixsize);
		  optix::Ray ray = optix::make_Ray( cam_pos, ray_direction, 0, 0.0f, RT_DEFAULT_MAX);
		  rtTrace( top_object, ray, prd );
		  result += prd.result;
	  }
	  prd.result = result / float(initial_samples);
  } else {	  
	  ray_direction = getViewRay(d.x, d.y, pixsize);
	  optix::Ray ray = optix::make_Ray( cam_pos, ray_direction, 0, 0.0f, RT_DEFAULT_MAX);
	  rtTrace( top_object, ray, prd );
	  prd.result = (output_buffer[launch_index]*(sample-1) + prd.result) / float(sample);
  }

  output_buffer[launch_index] = prd.result;
}

RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  //rtPrintf( "Exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
  printf( "Exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
  rtPrintExceptionDetails();
  output_buffer[launch_index] = bad_color;
}
